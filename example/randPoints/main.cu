#include <stdio.h>
#include <hip/hip_runtime.h>
#include ""
#include "Mesh.h"
#include "DeviceHostVector.h"
#include "bv.h"
#include "atomicFunctions.cuh"
#include "lbvh.h"


using namespace CXE;

//#include "BvhUtils.cuh"
int main() {
    const int N =  100000;
    const float R = 0.001f;

    printf("Generating Data...\n");
    DeviceHostVector<AABB> aabbs;
    aabbs.Allocate(N);
    srand(1);
    for (size_t i = 0; i < N; i++) {
        vec3f points = vec3f(rand() / (float)RAND_MAX, rand() / (float)RAND_MAX, rand() / (float)RAND_MAX);
        //vec3f points = vec3f(i*R*1.1, i*R*1.1, i*R*1.1);
        aabbs.GetHost()[i] = AABB(points.x - R , points.y - R, points.z - R,
            points.x + R, points.y + R, points.z + R);
    }

    //float temp = 0.194304f;
    //uint a = *reinterpret_cast<uint*>(&temp);
    //
    //for (int bit = 31;bit >= 0;bit--) {
	//	printf("%d", (a>>bit)&1);
    //}
    //
    //printf("\n");
    //
	//a &= 0x7F800000; // Clear the sign bit
	//a |= 0x00002000; // Set the exponent to 1
    //for (int bit = 31;bit >= 0;bit--) {
    //    printf("%d", (a >> bit) & 1);
    //}
    //
    //temp = *reinterpret_cast<float*>(&a);
	//printf("\nCleared sign bit: %.10f\n", temp);

    aabbs.ReadToDevice();
    Bvh A;
    A._type = 6;
    A.setup(N, N, N - 1);
    A.build(aabbs.GetDevice());
    A.query(aabbs.GetDevice(), aabbs.GetSize(),true);
    return 0;
}

