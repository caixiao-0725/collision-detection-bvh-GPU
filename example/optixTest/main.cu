﻿#include "optixLauncher.h"
#include <iostream>
#include <iomanip>
#include <hip/hip_runtime.h>

#include <sstream>
#include <fstream>
#include <vector>

#include "origin.h"
#include "DeviceHostVector.h"
#include "common.h"

using namespace CXE;

class Mesh {
public:
	DeviceHostVector<vec3f> points;
	DeviceHostVector<vec3u> faces;

	void readObj(const char* filename) {
        std::ifstream in;
        in.open(filename, std::ifstream::in); // 打开.obj文件
        if (in.fail())
            return;
        std::string line;
        while (!in.eof())
        {                           // 没有到文件末尾的话
            std::getline(in, line); // 读入一行
            std::istringstream iss(line.c_str());
            char trash;
            if (!line.compare(0, 2, "v "))
            { // 如果这一行的前两个字符是“v ”的话，代表是顶点数据
                iss >> trash;
                vec3f v; // 读入顶点坐标
                for (int i = 0; i < 3; i++)
                    iss >> v.raw[i];
                points.GetHost().push_back(v); // 加入顶点集
            }
            else if (!line.compare(0, 2, "f "))
            { // 如果这一行的前两个字符是“f ”的话，代表是面片数据
                vec3u v;
                int iuv, idx; // idx是顶点索引，itrash用来读我们暂时用不到的纹理坐标和法线向量
                iss >> trash;
                int i = 0;
                while (iss >> idx)
                {          // 读取x/x/x格式
                    idx--; // all indices start at 1, not 0
                    v.raw[i] = idx;
                    ; // 加入该面片的顶点集
                    i++;
                }
                faces.GetHost().push_back(v); // 把该面片加入模型的面片集
            }
        }

        points.SyncHostSize();
        faces.SyncHostSize();

        points.Allocate(points.GetSize());
        faces.Allocate(faces.GetSize());

        points.ReadToDevice();
        faces.ReadToDevice();
	}
};


int main() {
    // edge
    DeviceHostVector<vec3f> edges_points;
    DeviceHostVector<vec2i> edges_indexs;

    edges_points.Allocate(2);
    edges_indexs.Allocate(1);

    edges_points.GetHost()[0] = vec3f(-0.01, 1, 0);
	edges_points.GetHost()[1] = vec3f(-0.01, -1, 0);

	edges_indexs.GetHost()[0] = vec2i(1, 0);

	edges_points.ReadToDevice();
	edges_indexs.ReadToDevice();
    // model;

	Mesh m_obstacle;

	std::string path = get_asset_path() + "plane/20.obj";
    m_obstacle.readObj(path.c_str());

	OptixLauncher temp;
	temp.init();

    uint32_t vertexStride = 3 * sizeof(float);
    uint32_t posOffset = 0;
    uint32_t vertexCount = m_obstacle.points.GetSize();
    uint32_t indexCount = m_obstacle.faces.GetSize()*3;
    float transform[3][4];
    temp.buildObstacle((void*)m_obstacle.points.GetDevice(), vertexStride, posOffset, vertexCount,
        (void*)m_obstacle.faces.GetDevice(), indexCount, transform);

    temp.launchForEdge((void*)edges_points.GetDevice(), (void*)edges_indexs.GetDevice(), edges_indexs.GetSize());

    HitResult result_cpu[20];
	hipMemcpy(result_cpu, temp.m_gpuHitResults, sizeof(HitResult) * 20, hipMemcpyDeviceToHost);

	return 0;
}