#include "lbvh.h"
#include "BvhUtils.cuh"
#include "hipcub/hipcub.hpp"
 

using namespace CXE;

int const K_THREADS = 256;

void Bvh::setup(int prim_size, int ext_node_size, int int_node_size) {

	_primSize = prim_size;
	_extSize = ext_node_size;
	_intSize = int_node_size;
	_extNodes.setup(_primSize, _extSize);
	_unsortedTks.setup(_intSize);

	switch (_type)
	{
	case 0:
		_intNodes.setup(_intSize);
		break;
	case 1:
		_intNodes.setup(_intSize);
		break;
	case 2:
		_mergeNodes.setup(_intSize);
		break;
	case 3:
		_intNodes.setup(_intSize);
		_stacklessMergeNodes.setup(_intSize);
		break;
	case 4:
		_intNodes.setup(_intSize);
		_stacklessMergeNodesV1.setup(2*_intSize+1);
	case 11:
		_mergeNodes.setup(_intSize);
		break;
	case 12:
		_mergeNodes.setup(_intSize);
		break;
	}

	d_keys32.Allocate(_primSize);
	d_vals.Allocate(_primSize);
	d_primMap.Allocate(_primSize);
	d_count.Allocate(_primSize);
	d_bv.Allocate(1);
	d_tkMap.Allocate(_primSize);
	d_offsetTable.Allocate(_primSize);

	_cpNum.Allocate(1);
	_cpRes.Allocate(_primSize*32);
	
}

void Bvh::build(const vec3f* vertices, const vec3i* faces) {

	int blockDim = K_THREADS;
	int gridDim = (_primSize + blockDim - 1) / blockDim;

	BvhUtils::calcMaxBVWarpShuffle << <dim3(gridDim, 1, 1), dim3(blockDim, 1, 1) >> > (_primSize, faces, vertices, d_bv);
	BvhUtils::calcMCs<<<dim3(gridDim, 1, 1), dim3(blockDim, 1, 1) >>>(_primSize, faces, vertices, d_bv, _extNodes._mtcode);
	checkThrustErrors(thrust::sequence(thrust::device, d_vals.GetDevice(), d_vals.GetDevice() + d_vals.GetSize()));
	checkThrustErrors(thrust::sort_by_key(thrust::device, _extNodes._mtcode.GetDevice(), _extNodes._mtcode.GetDevice() + _extNodes._mtcode.GetSize(), d_vals.GetDevice()));
	BvhUtils::calcInverseMapping << <dim3(gridDim, 1, 1), dim3(blockDim, 1, 1) >> > (_primSize, d_vals, d_primMap);

	BvhUtils::buildPrimitives << < gridDim, blockDim >> > (_primSize,lvs()._idx, lvs()._box, d_primMap, faces, vertices);

	// build external nodes
    lvs().buildExtNodes(_primSize);
	lvs().calcSplitMetrics(extSize());

	// build internal nodes
	tks().clearIntNodes(_primSize-1);
	BvhUtils::buildIntNodes << < dim3(gridDim, 1, 1), dim3(blockDim, 1, 1) >> > (_primSize, d_count, lvs()._lca, lvs()._metric, lvs()._par, lvs()._mark, lvs()._box,
		untks()._rc, untks()._lc, untks()._rangey, untks()._rangex, untks()._mark, untks()._box, untks()._flag, untks()._par);
	
}

void Bvh::reorderIntNodes() {

	checkThrustErrors(thrust::exclusive_scan(thrust::device,d_count.GetDevice(), d_count.GetDevice() + extSize(), d_offsetTable.GetDevice()));
	const int blockDim = K_THREADS;
	const int gridDim = (extSize() + blockDim - 1) / blockDim;
	BvhUtils::calcIntNodeOrders << < gridDim, blockDim >> > (extSize(), untks()._lc, lvs()._lca, d_count, d_offsetTable, d_tkMap);

	checkThrustErrors(thrust::fill(thrust::device, lvs()._lca.GetDevice() + extSize(),
		lvs()._lca.GetDevice() + extSize() + 1, -1));
	
	
	BvhUtils::updateBvhExtNodeLinks << <gridDim, blockDim >> > (extSize(),d_tkMap, lvs()._lca, lvs()._par);
	const int intGridDim = (intSize() + blockDim - 1) / blockDim;
	switch (_type)
	{
	case 0:
		BvhUtils::reorderIntNode << <intGridDim, blockDim >> > (intSize(), d_tkMap, untks()._lc, untks()._rc, untks()._mark, untks()._par, untks()._rangex, untks()._rangey, untks()._box,
			tks()._lc, tks()._rc, tks()._mark, tks()._par, tks()._rangex, tks()._rangey, tks()._box);
		break;
	case 1:
		BvhUtils::reorderIntNode << <intGridDim, blockDim >> > (intSize(), d_tkMap, untks()._lc, untks()._rc, untks()._mark, untks()._par, untks()._rangex, untks()._rangey, untks()._box,
			tks()._lc, tks()._rc, tks()._mark, tks()._par, tks()._rangex, tks()._rangey, tks()._box);
		break;
	case 2:
		BvhUtils::reorderMergeNode << <intGridDim, blockDim >> > (intSize(), d_tkMap, _mergeNodes._nodes, lvs()._box, untks()._box, untks()._lc, untks()._rc, untks()._mark, untks()._par);
		break;
	case 3:
		BvhUtils::reorderIntNodeSoa << <intGridDim, blockDim >> > (intSize(), d_tkMap, untks()._lc, untks()._rc, untks()._mark, untks()._par, untks()._rangex, untks()._rangey, untks()._box,
			tks()._lc, tks()._rc, tks()._mark, tks()._par, _stacklessMergeNodes._nodes);
		break;
	case 4:
		BvhUtils::reorderIntNodeV1 << <intGridDim, blockDim >> > (intSize(), d_tkMap, lvs()._lca,lvs()._box, 
			untks()._lc, untks()._mark, untks()._rangey, untks()._box,
			_stacklessMergeNodesV1._nodes);
		
		break;
	default:
		break;
	}
	
}


void Bvh::build(const AABB* boxs) {


	int blockDim = K_THREADS;
	int gridDim = (_primSize + blockDim - 1) / blockDim;

	BvhUtils::calcMaxBVFromBox << <dim3(gridDim, 1, 1), dim3(blockDim, 1, 1) >> > (_primSize, boxs, d_bv);
	BvhUtils::calcMCsFromBox << <dim3(gridDim, 1, 1), dim3(blockDim, 1, 1) >> > (_primSize, boxs, d_bv, lvs()._mtcode);

	checkThrustErrors(thrust::sequence(thrust::device, d_vals.GetDevice(), d_vals.GetDevice() + d_vals.GetSize()));
	checkThrustErrors(thrust::sort_by_key(thrust::device, lvs()._mtcode.GetDevice(), lvs()._mtcode.GetDevice() + lvs()._mtcode.GetSize(), d_vals.GetDevice()));
	BvhUtils::calcInverseMapping << <dim3(gridDim, 1, 1), dim3(blockDim, 1, 1) >> > (_primSize, d_vals, d_primMap);

	BvhUtils::buildPrimitivesFromBox << < gridDim, blockDim >> > (_primSize, lvs()._idx, lvs()._box, d_primMap, boxs);
	if (_type > 10) {
		BvhUtils::lbvhBuildInternalKernel << < gridDim, blockDim >> > (mgs()._nodes, lvs()._par, lvs()._mtcode, lvs()._idx, _primSize);

		mgs().clearFlags();
		BvhUtils::mergeNodeRefit << < gridDim, blockDim >> > (mgs()._nodes, lvs()._par, lvs()._box, lvs()._idx, mgs()._flags, extSize());
	}
	else {
		// build external nodes
		lvs().buildExtNodes(_primSize);
		lvs().calcSplitMetrics(extSize());
	
		// build internal nodes
		untks().clearIntNodes(_primSize - 1);
		BvhUtils::buildIntNodes << < dim3(gridDim, 1, 1), dim3(blockDim, 1, 1) >> > (_primSize, d_count, lvs()._lca, lvs()._metric, lvs()._par, lvs()._mark, lvs()._box,
			untks()._rc, untks()._lc, untks()._rangey, untks()._rangex, untks()._mark, untks()._box,untks()._flag, untks()._par);
		reorderIntNodes();
	}
	//refit ?
	//BvhUtils::refitIntNode << <(extSize() + 255) / 256, 256 >> > (extSize(), lvs()._par, tks()._par, tks()._flag, tks()._lc, tks()._rc, tks()._mark, lvs()._box, tks()._box);
}


void Bvh::query(const AABB* boxs, const uint num,bool self) {
	int blockDim = K_THREADS;
	int gridDim = (num + blockDim - 1) / blockDim;
	_stacklessMergeNodesV1._nodes.ReadToHost();
	if (self) {
		switch (_type)
		{
		case 0:
			BvhUtils::pureBvhStacklessCD<true> << <gridDim, blockDim >> > (num, boxs, lvs()._par, lvs()._idx, lvs()._box, lvs()._lca,
				tks()._box, tks()._rangex, tks()._rangey,
				_cpNum, _cpRes
				);
			break;
		case 1:
			BvhUtils::pureBvhStackCD<true> << <gridDim, blockDim >> > (num, boxs, lvs()._par, lvs()._idx, lvs()._box, lvs()._lca,
				tks()._box, tks()._lc, tks()._rc, tks()._mark,
				_cpNum, _cpRes
				);
			break;
		case 2:
			BvhUtils::pureMergeBvhStackCD<true> << <gridDim, blockDim >> > (num, boxs,
				_mergeNodes._nodes, lvs()._idx,
				_cpNum, _cpRes);
			break;
		case 3:
			BvhUtils::AosBvhStacklessCD<true> << <gridDim, blockDim >> > (num, boxs, lvs()._par, lvs()._idx, lvs()._box, lvs()._lca,
				_stacklessMergeNodes._nodes,
				_cpNum, _cpRes
				);
			break;
		case 4:
			BvhUtils::AosBvhStacklessCDV1<true> << <gridDim, blockDim >> > (num, boxs, intSize(),lvs()._idx,
				_stacklessMergeNodesV1._nodes,
				_cpNum, _cpRes
				);
			break;
		case 11:
			BvhUtils::pureMergeBvhStackCD<true> << <gridDim, blockDim >> > (num, boxs,
				_mergeNodes._nodes, lvs()._idx,
				_cpNum, _cpRes);
			break;
		case 12:
			BvhUtils::pureMergeBvhStackSortElementCD<true> << <gridDim, blockDim >> > (num, boxs,
				_mergeNodes._nodes, lvs()._idx,
				_cpNum, _cpRes);
			break;
		}
	}
	else {
		switch (_type)
		{
		case 0:
			BvhUtils::pureBvhStacklessCD<false> << <gridDim, blockDim >> > (num, boxs, lvs()._par, lvs()._idx, lvs()._box, lvs()._lca,
				tks()._box, tks()._rangex, tks()._rangey,
				_cpNum, _cpRes
				);
			break;
		case 1:
			BvhUtils::pureBvhStackCD<false> << <gridDim, blockDim >> > (num, boxs, lvs()._par, lvs()._idx, lvs()._box, lvs()._lca,
				tks()._box, tks()._lc, tks()._rc, tks()._mark,
				_cpNum, _cpRes
				);
			break;
		case 2:
			BvhUtils::pureMergeBvhStackCD<false> << <gridDim, blockDim >> > (num, boxs,
				_mergeNodes._nodes, lvs()._idx,
				_cpNum, _cpRes);
			break;
		case 3:
			BvhUtils::AosBvhStacklessCD<false> << <gridDim, blockDim >> > (num, boxs, lvs()._par, lvs()._idx, lvs()._box, lvs()._lca,
				_stacklessMergeNodes._nodes,
				_cpNum, _cpRes
				);
			break;
		case 4:
			BvhUtils::AosBvhStacklessCDV1<false> << <gridDim, blockDim >> > (num, boxs, intSize(), lvs()._idx,
				_stacklessMergeNodesV1._nodes,
				_cpNum, _cpRes
				);
			break;
		case 11:
			BvhUtils::pureMergeBvhStackCD<false> << <gridDim, blockDim >> > (num, boxs,
				_mergeNodes._nodes, lvs()._idx,
				_cpNum, _cpRes);
			break;
		case 12:
			BvhUtils::pureMergeBvhStackSortElementCD<false> << <gridDim, blockDim >> > (num, boxs,
				_mergeNodes._nodes, lvs()._idx,
				_cpNum, _cpRes);
			break;
		}
	}
	


	_cpNum.ReadToHost();
	_cpRes.ReadToHost();

	printf("%d\n", _cpNum.GetHost()[0]);
}