#include "IntNode.h"

using namespace CXE;

void IntNodeArray::setup(const int intSize) {
	_intSize = intSize;
	hipMalloc((void**)&_attribs[FLAG], sizeof(uint) * intSize);
	hipMalloc((void**)&_attribs[LC], sizeof(int) * intSize);
	hipMalloc((void**)&_attribs[RC], sizeof(int) * intSize);
	hipMalloc((void**)&_attribs[PAR], sizeof(int) * intSize);
	hipMalloc((void**)&_attribs[MARK], sizeof(uint) * intSize);
	hipMalloc((void**)&_attribs[RANGEX], sizeof(uint) * intSize);
	hipMalloc((void**)&_attribs[RANGEY], sizeof(uint) * intSize);
}

void IntNodeArray::clearIntNodes(int size) {
	hipMemset(_attribs[FLAG], 0, sizeof(uint) * size);
}