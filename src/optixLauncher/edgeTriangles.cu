#include "hip/hip_runtime.h"
#include <assert.h>
#include <hip/hip_vector_types.h>

#include "optix.h"
#include "optix_types.h"

#include "edgeTriangles.h"

#define float3_as_uints(u) __float_as_uint(u.x), __float_as_uint(u.y), __float_as_uint(u.z)

extern "C"
{
	__constant__ Params params;
}


extern "C" __global__ void __miss__ms()
{
	optixSetPayload_4(0);  // hit
}

static __forceinline__ __device__ vec3f optixGetObjectRayO()
{
	float f0, f1, f2;
	asm("call (%0), _optix_get_object_ray_origin_x, ();" : "=f"(f0) : );
	asm("call (%0), _optix_get_object_ray_origin_y, ();" : "=f"(f1) : );
	asm("call (%0), _optix_get_object_ray_origin_z, ();" : "=f"(f2) : );
	return vec3f(f0, f1, f2);
}

static __forceinline__ __device__ vec3f optixGetObjectRayD()
{
	float f0, f1, f2;
	asm("call (%0), _optix_get_object_ray_direction_x, ();" : "=f"(f0) : );
	asm("call (%0), _optix_get_object_ray_direction_y, ();" : "=f"(f1) : );
	asm("call (%0), _optix_get_object_ray_direction_z, ();" : "=f"(f2) : );
	return vec3f(f0, f1, f2);
}

static __forceinline__ __device__ void trace_sphere(OptixTraversableHandle handle,
	float3 ray_origin,
	float3 ray_direction,
	float tmin,
	float tmax,
	HitResult* prd)
{
	unsigned int p[2];
	optixTrace(handle,
		ray_origin,
		ray_direction,
		tmin,
		tmax,
		0.0f,  // rayTime
		OptixVisibilityMask(255),
		OPTIX_RAY_FLAG_ENFORCE_ANYHIT | OPTIX_RAY_FLAG_DISABLE_CLOSESTHIT,//OPTIX_RAY_FLAG_NONE,
		0,  // SBT offset
		0,  // SBT stride
		0,  // missSBTIndex
		p[0], p[1]);


	prd->edgeIndex = (int)p[0];
	prd->faceIndex = (int)p[1];
}

extern "C" __global__ void __raygen__rg_edge()
{
	const uint3 idx = optixGetLaunchIndex();
	const uint3 dim = optixGetLaunchDimensions();

	unsigned int index = idx.x;

	float3 v0 = params.vertexs[params.edgeIndex[index].x];
	float3 v1 = params.vertexs[params.edgeIndex[index].y];

	const float3 d = float3{v1.x-v0.x,v1.y-v0.y,v1.z-v0.z};
	const float length = sqrtf(d.x * d.x + d.y * d.y + d.z * d.z);
	trace_sphere(params.handle,
		v0,
		d,
		0.00f,           // tmin
		length        ,  // tmax  //default 1.2
		&params.hitResults[index]);
}

extern "C" __global__ void __anyhit__ch()
{
	uint3 idx = optixGetLaunchIndex();  // ray's index
	const uint3 dim = optixGetLaunchDimensions();
	unsigned int ray_idx = idx.x;
	optixSetPayload_0(ray_idx);

	const unsigned int prim_idx = optixGetPrimitiveIndex();
	optixSetPayload_1(prim_idx);

	printf("%d   %d\n", ray_idx, prim_idx);

	optixIgnoreIntersection();
}


extern "C" __global__ void __closesthit__ch()
{
	uint3 idx = optixGetLaunchIndex();  // ray's index
	const uint3 dim = optixGetLaunchDimensions();
	unsigned int ray_idx = idx.x;
	optixSetPayload_0(ray_idx);

	const unsigned int prim_idx = optixGetPrimitiveIndex();
	optixSetPayload_1(prim_idx);

	printf("%d   %d\n", ray_idx, prim_idx);

}


extern "C" __global__ void __intersection__is()
{
	const vec3f e0 = optixGetObjectRayO();
	const vec3f e1 = optixGetObjectRayD() + e0;

	const unsigned int prim_idx = optixGetPrimitiveIndex();

	vec3i face = params.indexBuffer[prim_idx];
	vec3f v0 = params.vertexBuffer[face.x];
	vec3f v1 = params.vertexBuffer[face.y];
	vec3f v2 = params.vertexBuffer[face.z];

	optixReportIntersection(
		0.0f,  // t
		0,     // rayFlags
		0u,  // barycentrics
		0u,  // primitive index
		0u   // geometry index
	);

	//printf("%d   %d\n", ray_idx, prim_idx);


}